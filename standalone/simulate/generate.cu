#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>

#include <TFile.h>
#include <TH1F.h>
#include <TRandom3.h>
#include <TString.h>
#include <TTree.h>
#include <Math/Vector3D.h>

#include "include/event/generate_event.h"
#include "include/calculator/target_energy_calculator.h"

using namespace ribll;

using elc::TargetEnergyCalculator;

constexpr double beam_mass = mass_14c;
constexpr double target_mass = mass_2h;
constexpr double fragment2_mass = mass_4he;
constexpr double target_thickness = 9.53;


__device__ void InelasticScattering(
	double beam_mass,
	double target_mass,
	double beam_energy,
	double excited_energy,
	double theta,
	double &exit_energy,
	double &recoil_energy,
	double &exit_angle,
	double &recoil_angle
) {
	double beam_momentum = sqrt(pow(beam_energy, 2.0) - pow(beam_mass, 2.0));
	double beta_mass_center = beam_momentum / (beam_energy + target_mass);
	double gamma_mass_center =
		1.0 / sqrt(1.0 - beta_mass_center * beta_mass_center);
	double reaction_energy = sqrt(
		(beam_energy + target_mass + beam_momentum)
		* (beam_energy + target_mass - beam_momentum)
	);
	// mass of exit particle
	double exit_mass = beam_mass + excited_energy;
	// mass of recoil particle
	double recoil_mass = target_mass;
	// momentum of exit particle or recoil particle in center of mass frame
	double exit_momentum_center =
		sqrt(
			(reaction_energy - exit_mass - recoil_mass)
			* (reaction_energy - exit_mass + recoil_mass)
			* (reaction_energy + exit_mass - recoil_mass)
			* (reaction_energy + exit_mass + recoil_mass)
		) / (2.0 * reaction_energy);
	// exit momentum parallel and vertical part
	double exit_momentum_center_parallel = exit_momentum_center * cos(theta);
	double exit_momentum_center_vertical = exit_momentum_center * sin(theta);
	// exit energy in c.m.
	double exit_energy_center = sqrt(
		exit_momentum_center * exit_momentum_center
		+ exit_mass * exit_mass
	);
	// exit energy in lab frame
	exit_energy = gamma_mass_center * exit_energy_center
		+ gamma_mass_center * beta_mass_center * exit_momentum_center_parallel;
	// exit momentum parallel part in lab frame
	double exit_momentum_parallel = gamma_mass_center * exit_momentum_center_parallel
		+ gamma_mass_center * beta_mass_center * exit_energy_center;
	// exit momentum vertical part in lab frame
	double exit_momentum_vertical = exit_momentum_center_vertical;
	// calculate exit angle in lab frame
	exit_angle = fabs(atan(exit_momentum_vertical / exit_momentum_parallel));
	exit_angle = exit_momentum_parallel > 0 ?
		exit_angle : pi - exit_angle;

	// p of recoil particle in center of mass frame
	double recoil_momentum_center = exit_momentum_center;
	// recoil momentum parallel and vertical part
	double recoil_momentum_center_parallel = -recoil_momentum_center * cos(theta);
	double recoil_momentum_center_vertical = -recoil_momentum_center * sin(theta);
	// recoil energy in c.m.
	double recoil_energy_center = sqrt(
		recoil_momentum_center * recoil_momentum_center
		+ recoil_mass * recoil_mass
	);
	// recoil energy in lab frame
	recoil_energy = gamma_mass_center * recoil_energy_center
		+ gamma_mass_center * beta_mass_center * recoil_momentum_center_parallel;
	// recoil momentum parallel part in lab frame
	double recoil_momentum_parallel =
		gamma_mass_center * recoil_momentum_center_parallel
		+ gamma_mass_center * beta_mass_center * recoil_energy_center;
	// recoil momentum vertical part in lab frame
	double recoil_momentum_vertical = recoil_momentum_center_vertical;
	// recoil angle in lab frame
	recoil_angle =
		fabs(atan(recoil_momentum_vertical / recoil_momentum_parallel));
	recoil_angle = recoil_momentum_parallel > 0 ?
		recoil_angle : pi - recoil_angle;
}


__device__ void BreakupReaction(
	double parent_mass,
	double fragment1_mass,
	double fragment2_mass,
	double parent_energy,
	double theta,
	double &fragment1_energy,
	double &fragment2_energy,
	double &fragment1_angle,
	double &fragment2_angle
) {
	// momentum of parent particle in lab frame
	double parent_momentum =
		sqrt(pow(parent_energy, 2.0) - pow(parent_mass, 2.0));
	// beta of center of mass
	double beta_center = parent_momentum / parent_energy;
	// gamma of center of mass
	double gamma_center = 1.0 / sqrt(1.0 - pow(beta_center, 2.0));
	// total energy in center of mass frame
	double parent_energy_center = parent_mass;
	// momentum of fragments in center of mass frame
	double fragment_momentum =
		sqrt(
			(parent_energy_center - fragment1_mass - fragment2_mass)
			* (parent_energy_center - fragment1_mass + fragment2_mass)
			* (parent_energy_center + fragment1_mass - fragment2_mass)
			* (parent_energy_center + fragment1_mass + fragment2_mass)
		) / (2.0 * parent_mass);

	// fragment1 momentum parallel part in c.m. frame
	double fragment1_momentum_center_parallel = fragment_momentum*cos(theta);
	// fragment1 momentum vertical part in c.m. frame
	double fragment1_momentum_center_vertical = fragment_momentum*sin(theta);
	// fragment1 energy in c.m. frame
	double fragment1_energy_center = sqrt(
		pow(fragment_momentum, 2.0) + pow(fragment1_mass, 2.0)
	);
	// fragment1 energy in lab frame
	fragment1_energy = gamma_center * fragment1_energy_center
		+ gamma_center * beta_center * fragment1_momentum_center_parallel;
	// fragment1 momentum parallel part in lab frame
	double fragment1_momentum_parallel =
		gamma_center * fragment1_momentum_center_parallel
		+ gamma_center * beta_center * fragment1_energy_center;
	// fragment1 momentum vertical part in lab frame
	double fragment1_momentum_vertical = fragment1_momentum_center_vertical;
	// fragment1 angle in lab frame
	fragment1_angle =
		fabs(atan(fragment1_momentum_vertical / fragment1_momentum_parallel));
	fragment1_angle = fragment1_momentum_parallel > 0 ?
		fragment1_angle : pi - fragment1_angle;

	// fragment2 momentum parallel part in c.m. frame
	double fragment2_momentum_center_parallel = -fragment_momentum*cos(theta);
	// fragment2 momentum vertical part in c.m. frame
	double fragment2_momentum_center_vertical = -fragment_momentum*sin(theta);
	// fragment2 energy in c.m. frame
	double fragment2_energy_center = sqrt(
		pow(fragment_momentum, 2.0) + pow(fragment2_mass, 2.0)
	);
	// fragment2 energy in lab frame
	fragment2_energy = gamma_center * fragment2_energy_center
		+ gamma_center * beta_center * fragment2_momentum_center_parallel;
	// fragment2 momentum parallel part in lab frame
	double fragment2_momentum_parallel =
		gamma_center * fragment2_momentum_center_parallel
		+ gamma_center * beta_center * fragment2_energy_center;
	// fragment2 momentum vertical part in lab frame
	double fragment2_momentum_vertical = fragment2_momentum_center_vertical;
	// fragment2 angle in lab frame
	fragment2_angle =
		fabs(atan(fragment2_momentum_vertical / fragment2_momentum_parallel));
	fragment2_angle = fragment2_momentum_parallel > 0 ?
		fragment2_angle : pi - fragment2_angle;

	return;
}


__device__ void Rotate(
	double parent_theta, double parent_phi,
	double fragment_theta, double fragment_phi,
	double &theta, double &phi
) {
	double x =
		sin(fragment_theta)*cos(fragment_phi)*cos(parent_theta)*cos(parent_phi)
		-sin(fragment_theta)*sin(fragment_phi)*sin(parent_phi)
		+cos(fragment_theta)*sin(parent_theta)*cos(parent_phi);
	double y =
		sin(fragment_theta)*cos(fragment_phi)*cos(parent_theta)*sin(parent_phi)
		+sin(fragment_theta)*sin(fragment_phi)*cos(parent_phi)
		+cos(fragment_theta)*sin(parent_theta)*sin(parent_phi);
	double z =
		-sin(fragment_theta)*cos(fragment_phi)*sin(parent_theta)
		+cos(fragment_theta)*cos(parent_theta);

	theta = fabs(atan(sqrt(pow(x, 2.0) + pow(y, 2.0)) / z));
	theta = z > 0 ? theta : pi - theta;
	phi = atan(y / x);
	if (x < 0) {
		phi = y > 0 ? phi + pi : phi - pi;
	}
	return;
}

__device__ void VectorMultiply(double3 &vec, double multiply) {
	vec.x *= multiply;
	vec.y *= multiply;
	vec.z *= multiply;
}

__global__ void React(GenerateEvent *event, unsigned int n ) {
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < n) {
		double beam_energy = beam_mass + event[index].beam_kinetic_in_target;
		double parent_energy, recoil_energy, parent_angle, recoil_angle;

		double parent_mass = beam_mass + event[index].beam_excited_energy;

		// inelastic scattering
		InelasticScattering(
			beam_mass, target_mass,
			beam_energy, event[index].beam_excited_energy,
			event[index].elastic_angle,
			parent_energy, recoil_energy, parent_angle, recoil_angle
		);
		event[index].parent_kinetic = parent_energy - parent_mass;
		event[index].recoil_kinetic_in_target = recoil_energy - target_mass;

		// rotate from beam frame to lab frame
		Rotate(
			event[index].beam_theta, event[index].beam_phi,
			parent_angle, event[index].parent_phi,
			event[index].parent_theta, event[index].parent_phi
		);
		Rotate(
			event[index].beam_theta, event[index].beam_phi,
			recoil_angle, event[index].recoil_phi,
			event[index].recoil_theta, event[index].recoil_phi
		);

		// get angle of parent relate to recoil
		double3 v_parent = {
			sin(parent_angle)*cos(event[index].parent_phi),
			sin(parent_angle)*sin(event[index].parent_phi),
			cos(parent_angle)
		};
		double v_parent_value =
			sqrt(pow(parent_energy, 2.0) - pow(parent_mass, 2.0))
			/ parent_energy;
		VectorMultiply(v_parent, v_parent_value);

		// recoil vector direction
		double3 v_recoil = {
			sin(recoil_angle)*cos(event[index].recoil_phi),
			sin(recoil_angle)*sin(event[index].recoil_phi),
			cos(recoil_angle)
		};
		double v_recoil_value =
			sqrt(pow(recoil_energy, 2.0) - pow(target_mass, 2.0))
			/ recoil_energy;
		VectorMultiply(v_recoil, v_recoil_value);

		double3 relative_v_parent_recoil = {
			v_parent.x - v_recoil.x,
			v_parent.y - v_recoil.y,
			v_parent.z - v_recoil.z
		};
		event[index].parent_recoil_angle =
			sqrt(pow(relative_v_parent_recoil.x, 2.0) + pow(relative_v_parent_recoil.y, 2.0))
			/ relative_v_parent_recoil.z;

		event[index].rz = 135.0;
		event[index].rr = event[index].rz * tan(event[index].recoil_theta);
		event[index].rx = event[index].rr * cos(event[index].recoil_phi)
			+ event[index].target_x;
		event[index].ry = event[index].rr * sin(event[index].recoil_phi)
			+ event[index].target_y;
		event[index].rr = sqrt(pow(event[index].rx, 2.0)
			+ pow(event[index].ry, 2.0));

		double fragment1_mass =
			mass_10be + event[index].fragment_excited_energy;

		double fragment1_energy, fragment2_energy;
		double fragment1_angle, fragment2_angle;
		BreakupReaction(
			parent_mass, fragment1_mass, fragment2_mass,
			parent_energy, event[index].breakup_angle,
			fragment1_energy, fragment2_energy,
			fragment1_angle, fragment2_angle
		);

		// rotate and get fragment 1 angle
		Rotate(
			event[index].parent_theta, event[index].parent_phi,
			fragment1_angle, event[index].fragment_phi_center,
			event[index].fragment_theta[0], event[index].fragment_phi[0]
		);
		// rotate and get fragment 2 angle
		Rotate(
			event[index].parent_theta, event[index].parent_phi,
			fragment2_angle, event[index].fragment_phi_center-pi,
			event[index].fragment_theta[1], event[index].fragment_phi[1]
		);
		// calculate kinetic energy for fragments
		event[index].fragment_kinetic_in_target[0] =
			fragment1_energy - fragment1_mass;
		event[index].fragment_kinetic_in_target[1] =
			fragment2_energy - fragment2_mass;


		// // get angle of parent relate to recoil
		// ROOT::Math::XYZVector v_fragment1(
		// 	sin(fragment1_angle)*cos(event[index].fragment_phi_center),
		// 	sin(fragment1_angle)*sin(event[index].fragment_phi_center),
		// 	cos(fragment1_angle)
		// );
		// v_fragment1 *=
		// 	sqrt(pow(fragment1_energy, 2.0) - pow(fragment1_mass, 2.0))
		// 	/ sqrt(pow(fragment1_energy, 2.0) + pow(fragment1_mass, 2.0));

		// ROOT::Math::XYZVector v_fragment2(
		// 	sin(fragment2_angle)*cos(event[index].fragment_phi_center-pi),
		// 	sin(fragment2_angle)*sin(event[index].fragment_phi_center-pi),
		// 	cos(fragment2_angle)
		// );
		// v_fragment2 *=
		// 	sqrt(pow(fragment2_energy, 2.0) - pow(fragment2_mass, 2.0))
		// 	/ sqrt(pow(fragment2_energy, 2.0) + pow(fragment2_mass, 2.0));

		// event[index].fragment_fragment_angle = (v_fragment1 - v_fragment2).Theta();
	}
}


__host__ void BeforeReact(
	int run,
	size_t index,
	size_t total,
	GenerateEvent &event,
	TRandom3 &generator,
	const TH1F &dist_scatter_angle,
	const TH1F &dist_breakup_angle,
	TargetEnergyCalculator &c14_target

) {
	// get reaction point x
		event.target_x = generator.Gaus(0.0, 6.0);
		// get reaction point y
		event.target_y = generator.Gaus(0.0, 6.0);
		// get beam trace at z = -800
		double beam_trace_x = generator.Gaus(0.0, 5.0);
		double beam_trace_y = generator.Gaus(0.0, 5.0);
		event.beam_phi = atan(beam_trace_y / beam_trace_x);
		if (beam_trace_x > 0) {
			event.beam_phi = beam_trace_y < 0
				? event.beam_phi + pi
				: event.beam_phi - pi;
		}
		event.beam_theta = atan(
			sqrt(pow(beam_trace_x, 2.0) + pow(beam_trace_y, 2.0)) / 800.0
		);

		// get beam kinetic energy
		event.beam_kinetic_before_target =
			generator.Gaus(389.5, 3.5);
		// reaction point depth
		event.depth = generator.Rndm();
		// consider energy loss in target
		event.beam_kinetic_in_target = c14_target.Energy(
			event.depth / cos(event.beam_theta),
			event.beam_kinetic_before_target
		);
		// get beam excited energy
		if (run == 0) {
			if (index < total / 3) {
				event.beam_excited_energy =
					12.0125 + generator.Rndm() * (20.0 - 12.0125) + 0.1;
			} else if (index < total / 3 * 2) {
				event.beam_excited_energy =
					12.0125 + 3.368
					+ generator.Rndm() * (40.0 - 12.0125 - 3.368) + 0.1;
			} else {
				event.beam_excited_energy =
					12.0125 + 6.179
					+ generator.Rndm() * (40.0 - 12.0125 - 6.179) + 0.1;
			}
		} else if (run == 1 || run == 2) {
			if (index < total / 3) {
				event.beam_excited_energy =
					12.02 + 0.2 * ((index / 10'000) % 100);
			} else if (index < total / 3 * 2) {
				event.beam_excited_energy =
					15.39 + 0.2 * ((index / 10'000) % 100);
			} else {
				event.beam_excited_energy =
					18.20 + 0.2 * ((index / 10'000) % 100);
			}
		}

		// get elastic angle theta
		event.elastic_angle = dist_scatter_angle.GetRandom();
		// get breakup angle phi
		event.parent_phi = generator.Rndm() * 2.0 * pi;
		event.recoil_phi = event.parent_phi - pi;

		// get fragment excited energy
		event.fragment_excited_energy = 0.0;
		event.fragment_state = 0;
		if (index >= total / 3 * 2) {
			event.fragment_excited_energy = 6.179;
			event.fragment_state = 2;
		} else if (index >= total / 3) {
			event.fragment_excited_energy = 3.368;
			event.fragment_state = 1;
		}
		// get breakup angle theta
		event.breakup_angle = dist_breakup_angle.GetRandom();
		// get breakup angle phi
		event.fragment_phi_center = generator.Rndm() * 2.0 * pi;
}


__host__ void AfterReact(
	GenerateEvent &event,
	TargetEnergyCalculator &he4_target,
	TargetEnergyCalculator &be10_target,
	TargetEnergyCalculator &h2_target
) {
	// consider energy loss of recoil particle in target
	event.recoil_kinetic_after_target = h2_target.Energy(
		(1.0-event.depth) / cos(event.recoil_theta),
		event.recoil_kinetic_in_target
	);

	if (event.recoil_kinetic_after_target < 0) {
		event.recoil_kinetic_after_target = 0.0;
	}

	// consider energy loss of fragment1 particle in target
	event.fragment_kinetic_after_target[0] = be10_target.Energy(
		(1.0-event.depth) / cos(event.fragment_theta[0]),
		event.fragment_kinetic_in_target[0]
	);
	// consider energy loss of fragment1 particle in target
	event.fragment_kinetic_after_target[1] = he4_target.Energy(
		(1.0-event.depth) / cos(event.fragment_theta[1]),
		event.fragment_kinetic_in_target[1]
	);

	for (size_t j = 0; j < 2; ++j) {
		event.fragment_z[j] = 100.0;
		event.fragment_r[j] =
			event.fragment_z[j] * tan(event.fragment_theta[j]);
		event.fragment_x[j] =
			event.fragment_r[j] * cos(event.fragment_phi[j])
			+ event.target_x;
		event.fragment_y[j] =
			event.fragment_r[j] * sin(event.fragment_phi[j])
			+ event.target_y;
		event.fragment_r[j] = sqrt(
			pow(event.fragment_x[j], 2.0)
			+ pow(event.fragment_y[j], 2.0)
		);
	}
}

void CheckCudaError(hipError_t error) {
    if (error != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(error) << "\n";
        exit(-1);
    }
}


int main(int argc, char **argv) {
	int run = 0;
	if (argc > 1) {
		run = atoi(argv[1]);
	}
	if (run < 0 || run > 2) {
		std::cout << "Usage: " << argv[0] << "[run]\n"
			<< "  run        run number, default is 0\n";
	}

	std::cout << "Start to simulate run " << run << "\n";

	TargetEnergyCalculator c14_target("14C", "CD2", target_thickness);
	TargetEnergyCalculator be10_target("10Be", "CD2", target_thickness);
	TargetEnergyCalculator he4_target("4He", "CD2", target_thickness);
	TargetEnergyCalculator h2_target("2H", "CD2", target_thickness);

	// generate file name
	TString generate_file_name = TString::Format(
		"%s%sgenerate-%04d.root",
		kGenerateDataPath,
		kSimulateDir,
		run
	);
	// generate file
	TFile generate_file(generate_file_name, "recreate");
	// scattering angle distribution in c.m. frame
	TH1F dist_scatter_angle("dsa", "distribution", 900, 0, pi);
	// breakup angle distribution in c.m. frame
	TH1F dist_breakup_angle("dba", "distribution", 900, 0, pi);
	// generate tree
	TTree tree("tree", "generated data");
	// generate data
	GenerateEvent event;
	// setup branches
	event.SetupOutput(&tree);

	// initialize generator
	TRandom3 generator(0);

	// parameter in scattering angle distribution
	constexpr double scatter_const = 55.0 / 180.0 * pi;
	// intialize distributions
	for (int i = 0; i < 900; ++i) {
		double x = dist_scatter_angle.GetBinCenter(i+1);
		dist_scatter_angle.SetBinContent(
			i+1, sin(x)*exp(-x/scatter_const)
		);
		dist_breakup_angle.SetBinContent(
			i+1, sin(x)
		);
	}

	constexpr size_t event_num = 3'000'000;
	// 1/100 of total event number
	size_t event100 = event_num / 100 + 1;
	GenerateEvent *generate_events = new GenerateEvent[event_num];
	// prepare events
	// show start
	printf("Preparing events   0%%");
	fflush(stdout);
	for (size_t i = 0; i < event_num; ++i) {
		// show process
		if (i % event100 == 0) {
			printf("\b\b\b\b%3lld%%", i / event100);
			fflush(stdout);
		}
		BeforeReact(
			run, i, event_num,
			generate_events[i], generator,
			dist_scatter_angle, dist_breakup_angle,
			c14_target
		);
	}
	// show finish
	printf("\b\b\b\b100%%\n");

	GenerateEvent *cuda_events;
	// allocate memory in GPU
	CheckCudaError(hipMalloc(&cuda_events, event_num * sizeof(GenerateEvent)));
	// copy events to GPU
	CheckCudaError(hipMemcpy(
		cuda_events, generate_events,
		event_num * sizeof(GenerateEvent),
		hipMemcpyHostToDevice
	));
	// simulate in GPU with CUDA
	React<<<(event_num+255)/256, 256>>>(cuda_events, event_num); 
	// copy from GPU
	CheckCudaError(hipMemcpy(
		generate_events, cuda_events,
		event_num * sizeof(GenerateEvent),
		hipMemcpyDeviceToHost
	));
	// free GPU memory
	CheckCudaError(hipFree(cuda_events));

	// after reaction
	// show start
	printf("Filling events   0%%");
	fflush(stdout);
	for (size_t i = 0; i < event_num; ++i) {
		// show process
		if (i % event100 == 0) {
			printf("\b\b\b\b%3lld%%", i / event100);
			fflush(stdout);
		}
		AfterReact(
			generate_events[i],
			he4_target, be10_target, h2_target
		);
		event = generate_events[i];
		tree.Fill();
	}
	// show finished
	printf("\b\b\b\b100%%\n");

	// save histograms
	dist_scatter_angle.Write();
	dist_breakup_angle.Write();
	// save tree
	tree.Write();
	// close files
	generate_file.Close();
	return 0;
}